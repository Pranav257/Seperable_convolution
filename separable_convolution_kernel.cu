
#include <hip/hip_runtime.h>
/* FIXME: Edit this file to complete the functionality of 2D separable 
 * convolution on the GPU. You may add additional kernel functions 
 * as necessary. 
 */

__global__ void convolve_rows_kernel_naive(float *result, float *input, float *kernel, int num_cols, int num_rows, int half_width)
{
    int i, i1;
    int j, j1, j2;
    int x, y;

    y = threadIdx.x + blockIdx.x * blockDim.x;

    for (x = 0; x < num_cols; x++) {

        j1 = x - half_width;
        j2 = x + half_width;

        if (j1 < 0) 
            j1 = 0;

        if (j2 >= num_cols) 
            j2 = num_cols - 1;

        i1 = j1 - x; 
        
        j1 = j1 - x + half_width; 

        j2 = j2 - x + half_width;

        result[y * num_cols + x] = 0.0;

        for(i = i1, j = j1; j <= j2; j++, i++){
            result[y * num_cols + x] += kernel[j] * input[y * num_cols + x + i];
        }
    }
}

__global__ void convolve_columns_kernel_naive(float *result, float *input, float *kernel, int num_cols, int num_rows, int half_width)
{
    int i, i1;
    int j, j1, j2;
    int x, y;

    y = threadIdx.x + blockIdx.x * blockDim.x;

    for(x = 0; x < num_cols; x++) {
        j1 = y - half_width;
        j2 = y + half_width;
        
        if (j1 < 0) 
            j1 = 0;

        if (j2 >= num_rows) 
            j2 = num_rows - 1;

        i1 = j1 - y; 
        
        j1 = j1 - y + half_width; 
        j2 = j2 - y + half_width;

        result[y * num_cols + x] = 0.0;

        for (i = i1, j = j1; j <= j2; j++, i++)
            result[y * num_cols + x] += 
                kernel[j] * input[y * num_cols + x + (i * num_cols)];
    }
}

__global__ void convolve_rows_kernel_optimized()
{
    return;
}

__global__ void convolve_columns_kernel_optimized()
{
    return;
}




